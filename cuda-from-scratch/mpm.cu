#include "hip/hip_runtime.h"
#include <cassert>
#include <fstream>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/tabulate.h>
#include <hip/hip_runtime.h>

#include "mpm.h"

#define IN_GRID(POS) (0 <= POS(0) && POS(0) < grid_bound_x && \
                      0 <= POS(1) && POS(1) < grid_bound_y && \
                      0 <= POS(2) && POS(2) < grid_bound_z)

__device__ float NX(const float& x)
{
    if (x < 1.0f)
        return 0.5f * (x * x * x) - (x * x) + (2.0f / 3.0f);

    if (x < 2.0f)
        return (-1.0f / 6.0f) * (x * x * x) + (x * x) - (2.0f * x) + (4.0f / 3.0f);

    return 0.0f;
}

__device__ float dNX(const float& x)
{
    float abs_x = fabs(x);

    if (abs_x < 1.0f)
    {
        return (1.5f * abs_x * x) - (2.0f * x);
    }
    else if (abs_x < 2.0f)
    {
        return -0.5f * (abs_x * x) + (2.0f * x) - (2.0f * x / abs_x);
    }
    else
    {
        return 0.0f;
    }
}

__device__ float weight(const Eigen::Vector3f& xpgp_diff)
{
    return NX(xpgp_diff(0)) * NX(xpgp_diff(1)) * NX(xpgp_diff(2));
}

__device__ Eigen::Vector3f gradientWeight(const Eigen::Vector3f& xpgp_diff)
{
    auto const& v = xpgp_diff;

    return (1.0f / particle_diameter) * Eigen::Vector3f(dNX(v(0)) * NX(fabs(v(1))) * NX(fabs(v(2))),
        NX(fabs(v(0))) * dNX(v(1)) * NX(fabs(v(2))),
        NX(fabs(v(0))) * NX(fabs(v(1))) * dNX(v(2)));
}

__device__ int getGridIndex(const Eigen::Vector3i& pos)
{
    return (pos(2) * grid_bound_y * grid_bound_x) + (pos(1) * grid_bound_x) + pos(0);
}

__device__ Eigen::Vector3f applyBoundaryCollision(const Eigen::Vector3f& position, const Eigen::Vector3f& velocity)
{
    float vn;
    Eigen::Vector3f vt, normal, ret(velocity);

    for (int i = 0; i < 3; i++)
    {
        bool collision = false;
        normal.setZero();

        if (position(i) <= box_boundary_1)
        {
            collision = true;
            normal(i) = 1.0f;
        }
        else if (position(i) >= box_boundary_2)
        {
            collision = true;
            normal(i) = -1.0f;
        }

        if (collision)
        {
            vn = ret.dot(normal);

            if (vn >= 0.0f)
                continue;

            for (int j = 0; j < 3; j++)
            {
                if (j != i)
                {
                    ret(j) *= stickiness;
                }
            }

            vt = ret - vn * normal;

            if (vt.norm() <= -friction * vn)
            {
                ret.setZero();
                return ret;
            }

            ret = vt + friction * vn * vt.normalized();
        }
    }

    return ret;
}

struct f
{
    __host__ __device__ Grid operator()(const int& idx) const
    {
        return Grid(Eigen::Vector3i(idx % grid_bound_x, idx % (grid_bound_x * grid_bound_y) / grid_bound_x, idx / (grid_bound_x * grid_bound_y)));
    }
};

MPMSolver::MPMSolver(const std::vector<Particle>& _particles)
{
    initialize(_particles);
}

__host__ MPMSolver::MPMSolver(const std::vector<Particle>& _particles, const std::vector<Grid>& _grids)
{
    particles.resize(_particles.size());
    grids.resize(_grids.size());

    thrust::copy(_particles.begin(), _particles.end(), particles.begin());
    thrust::copy(_grids.begin(), _grids.end(), grids.begin());
}

__host__ void MPMSolver::perform_initial_transfer()
{
    Grid* grid_ptr = thrust::raw_pointer_cast(&grids[0]);

    auto ff = [=] __device__(Particle & p)
    {
        float h_inv = 1.0f / particle_diameter;
        Eigen::Vector3i pos((p.position * h_inv).cast<int>());

        for (int z = -G2P; z <= G2P; z++)
        {
            for (int y = -G2P; y <= G2P; y++)
            {
                for (int x = -G2P; x <= G2P; x++)
                {
                    auto _pos = pos + Eigen::Vector3i(x, y, z);
                    if (!IN_GRID(_pos)) continue;

                    Eigen::Vector3f diff = (p.position - (_pos.cast<float>() * particle_diameter)) * h_inv;
                    int grid_idx = getGridIndex(_pos);
                    float mi = p.mass * weight(diff.cwiseAbs());
                    atomicAdd(&(grid_ptr[grid_idx].mass), mi);
                }
            }
        }
    };

    thrust::for_each(thrust::device, particles.begin(), particles.end(), ff);
}

__host__ void MPMSolver::initialize(std::vector<Particle> const& _particles)
{
    particles.resize(_particles.size());
    thrust::copy(_particles.begin(), _particles.end(), particles.begin());

    grids.resize(grid_bound_x * grid_bound_y * grid_bound_z);
    thrust::tabulate(
        thrust::device,
        grids.begin(),
        grids.end(),
        f()
    );
}

void MPMSolver::reset_grid()
{
    thrust::for_each(
        thrust::device,
        grids.begin(),
        grids.end(),
        [=] __device__(Grid & g)
    {
        g.reset();
    }
    );
}

__host__ void MPMSolver::transfer_data()
{
    Grid* grid_ptr = thrust::raw_pointer_cast(&grids[0]);

    auto ff = [=] __device__(Particle & p)
    {
        float constexpr h_inv = 1.0f / particle_diameter;
        Eigen::Vector3i const pos((p.position * h_inv).cast<int>());
        Eigen::Matrix3f const volume_stress = -1.0f * p.energy_derivative();

        for (int z = -G2P; z <= G2P; z++)
        {
            for (int y = -G2P; y <= G2P; y++)
            {
                for (int x = -G2P; x <= G2P; x++)
                {
                    auto _pos = pos + Eigen::Vector3i(x, y, z);
                    if (!IN_GRID(_pos)) continue;

                    Eigen::Vector3f diff = (p.position - (_pos.cast<float>() * particle_diameter)) * h_inv;
                    auto gw = gradientWeight(diff);
                    int grid_idx = getGridIndex(_pos);

                    Eigen::Vector3f f = volume_stress * gw;

                    float mi = p.mass * weight(diff.cwiseAbs());
                    atomicAdd(&(grid_ptr[grid_idx].mass), mi);
                    atomicAdd(&(grid_ptr[grid_idx].velocity(0)), p.velocity(0) * mi);
                    atomicAdd(&(grid_ptr[grid_idx].velocity(1)), p.velocity(1) * mi);
                    atomicAdd(&(grid_ptr[grid_idx].velocity(2)), p.velocity(2) * mi);
                    atomicAdd(&(grid_ptr[grid_idx].force(0)), f(0));
                    atomicAdd(&(grid_ptr[grid_idx].force(1)), f(1));
                    atomicAdd(&(grid_ptr[grid_idx].force(2)), f(2));
                }
            }
        }
    };

    thrust::for_each(thrust::device, particles.begin(), particles.end(), ff);
}

__host__ void MPMSolver::compute_volumes()
{
    Grid* const grid_ptr = thrust::raw_pointer_cast(&grids[0]);

    auto ff = [=] __device__(Particle & p) {
        float h_inv = 1.0f / particle_diameter;
        Eigen::Vector3i pos((p.position * h_inv).cast<int>());
        float p_density = 0.0f;
        float inv_grid_volume = h_inv * h_inv * h_inv;

        for (int z = -G2P; z <= G2P; z++)
        {
            for (int y = -G2P; y <= G2P; y++)
            {
                for (int x = -G2P; x <= G2P; x++)
                {
                    auto _pos = pos + Eigen::Vector3i(x, y, z);
                    if (!IN_GRID(_pos)) continue;

                    Eigen::Vector3f diff = (p.position - (_pos.cast<float>() * particle_diameter)) * h_inv;
                    int grid_idx = getGridIndex(_pos);
                    p_density += grid_ptr[grid_idx].mass * inv_grid_volume * weight(diff.cwiseAbs());
                }
            }
        }

        p.volume = p.mass / p_density;
    };

    thrust::for_each(thrust::device, particles.begin(), particles.end(), ff);
}

__host__ void MPMSolver::update_velocities()
{
    thrust::for_each(
        thrust::device,
        grids.begin(),
        grids.end(),
        [=] __device__(Grid & g) {
        g.update_velocity();
    }
    );
}

__host__ void MPMSolver::body_collisions()
{
    thrust::for_each(
        thrust::device,
        grids.begin(),
        grids.end(),
        [=] __device__(Grid & g) {
        g.velocity_star = applyBoundaryCollision((g.idx.cast<float>() * particle_diameter) + (step * g.velocity_star), g.velocity_star);
    }
    );
}

__host__ void MPMSolver::update_deformation_gradient()
{
    Grid const* grid_ptr = thrust::raw_pointer_cast(&grids[0]);

    auto compute_velocity_gradient = [=] __device__(const Particle & p) -> Eigen::Matrix3f
    {
        float h_inv = 1.0f / particle_diameter;
        Eigen::Vector3i pos((p.position * h_inv).cast<int>());
        Eigen::Matrix3f velocity_gradient(Eigen::Matrix3f::Zero());

        for (int z = -G2P; z <= G2P; z++)
        {
            for (int y = -G2P; y <= G2P; y++)
            {
                for (int x = -G2P; x <= G2P; x++)
                {
                    auto _pos = pos + Eigen::Vector3i(x, y, z);
                    if (!IN_GRID(_pos)) continue;

                    Eigen::Vector3f diff = (p.position - (_pos.cast<float>() * particle_diameter)) * h_inv;
                    Eigen::Vector3f gw = gradientWeight(diff);
                    int grid_idx = getGridIndex(_pos);

                    velocity_gradient += grid_ptr[grid_idx].velocity_star * gw.transpose();
                }
            }
        }

        return velocity_gradient;
    };

    thrust::for_each(
        thrust::device,
        particles.begin(),
        particles.end(),
        [=] __device__(Particle & p) {
        auto const velocity_gradient = compute_velocity_gradient(p);
        p.update_deformation_gradient(velocity_gradient);
    }
    );
}

__host__ void MPMSolver::update_particle_velocities()
{
    Grid const* grid_ptr = thrust::raw_pointer_cast(&grids[0]);

    auto compute_velocity = [=] __device__(const Particle & p) -> thrust::pair<Eigen::Vector3f, Eigen::Vector3f> {
        float constexpr h_inv = 1.0f / particle_diameter;
        Eigen::Vector3i const pos((p.position * h_inv).cast<int>());

        Eigen::Vector3f velocity_pic(Eigen::Vector3f::Zero()),
            velocity_flip(p.velocity);

        for (int z = -G2P; z <= G2P; z++) {
            for (int y = -G2P; y <= G2P; y++) {
                for (int x = -G2P; x <= G2P; x++) {
                    auto _pos = pos + Eigen::Vector3i(x, y, z);
                    if (!IN_GRID(_pos)) continue;

                    Eigen::Vector3f diff = (p.position - (_pos.cast<float>() * particle_diameter)) * h_inv;
                    int const grid_idx = getGridIndex(_pos);
                    float w = weight(diff.cwiseAbs());
                    auto grid = grid_ptr[grid_idx];
                    velocity_pic += grid.velocity_star * w;
                    velocity_flip += (grid.velocity_star - grid.velocity) * w;
                }
            }
        }

        return thrust::make_pair(velocity_pic, velocity_flip);
    };

    thrust::for_each(
        thrust::device,
        particles.begin(),
        particles.end(),
        [=] __device__(Particle & p) {
        auto const velocity_result = compute_velocity(p);
        p.update_velocity(velocity_result.first, velocity_result.second);
    }
    );
}

__host__ void MPMSolver::particle_body_collisions()
{
    thrust::for_each(
        thrust::device,
        particles.begin(),
        particles.end(),
        [=] __device__(Particle & p) {
        p.velocity = applyBoundaryCollision(p.position + step * p.velocity, p.velocity);
    }
    );
}

__host__ void MPMSolver::update_particle_positions()
{
    thrust::for_each(
        thrust::device,
        particles.begin(),
        particles.end(),
        [=] __device__(Particle & p) {
        p.update_position();
    }
    );
}

__host__ void MPMSolver::simulate()
{
    reset_grid();
    if (transfer_first_time)
    {
        perform_initial_transfer();
        compute_volumes();
        transfer_first_time = false;
    }
    else
    {
        transfer_data();
    }
    update_velocities();
    body_collisions();
    update_deformation_gradient();
    update_particle_velocities();
    particle_body_collisions();
    update_particle_positions();
}

__host__ void MPMSolver::bind_gl_buffer(const GLuint buffer)
{
    hipError_t ret;
    ret = hipGraphicsGLRegisterBuffer(&vbo_resource, buffer, cudaGraphicsMapFlagsWriteDiscard);
    assert(ret == hipSuccess);
}

__host__ void MPMSolver::write_gl_buffer()
{
    hipError_t ret;
    float4* bufptr;
    size_t size;

    ret = hipGraphicsMapResources(1, &vbo_resource, NULL);
    assert(ret == hipSuccess);
    ret = hipGraphicsResourceGetMappedPointer((void**)&bufptr, &size, vbo_resource);
    assert(ret == hipSuccess);

    assert(bufptr != nullptr && size >= particles.size() * sizeof(float4));
    thrust::transform(
        thrust::device,
        particles.begin(),
        particles.end(),
        bufptr,
        [=] __device__(Particle & p) -> float4 {
        return make_float4(5.0 * p.position(0) - 2.5, 5.0 * p.position(1), 5.0 * p.position(2) - 2.5, 1.0);
    }
    );

    ret = hipGraphicsUnmapResources(1, &vbo_resource, NULL);
    assert(ret == hipSuccess);
}

__host__ void MPMSolver::write_to_file(const std::string& filename)
{
    std::ofstream output(filename, std::ios::binary | std::ios::out);
    int num_particles = particles.size();
    float min_bound_x = 0, max_bound_x = grid_bound_x;
    float min_bound_y = 0, max_bound_y = grid_bound_y;
    float min_bound_z = 0, max_bound_z = grid_bound_z;

    output.write(reinterpret_cast<char*>(&num_particles), sizeof(int));
    output.write(reinterpret_cast<char*>(&min_bound_x), sizeof(float));
    output.write(reinterpret_cast<char*>(&max_bound_x), sizeof(float));
    output.write(reinterpret_cast<char*>(&min_bound_y), sizeof(float));
    output.write(reinterpret_cast<char*>(&max_bound_y), sizeof(float));
    output.write(reinterpret_cast<char*>(&min_bound_z), sizeof(float));
    output.write(reinterpret_cast<char*>(&max_bound_z), sizeof(float));

    thrust::copy(
        particles.begin(),
        particles.end(),
        std::ostream_iterator<Particle>(output)
    );

    output.close();
}
